#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <cmath>

// helper functions
template <typename scalar_t>
__device__ __forceinline__ scalar_t relu(scalar_t z) {
  return fmax(0.0, z);
}

// relu test kernel

template <typename scalar_t>
__global__ void nl_relu_kernel(
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> output) {
  //batch index
  const int n = blockIdx.x;
  //channel index
  const int c = threadIdx.x;
  //height index
  const int w = threadIdx.y;
  //width index
  const int h = threadIdx.z;
  if (n < input.size(0) && c < input.size(1) && w < input.size(2) && h < input.size(3)){
    output[n][c][w][h] = relu(input[n][c][w][h]);
  }
}

// relu test kernel wrapper function

std::vector<torch::Tensor> nl_relu_cuda(
  torch::Tensor input){
    auto input_size = input.sizes();
    const int num_batch   = input_size[0];
    const int num_channel = input_size[1];
    const int height = input_size[2];
    const int width = input_size[3];

    //std::cout << "(N,C,H,W) = (" << num_batch << ", " << num_channel << ", "
    //<< height << ", " << width << ")" << std::endl;

    //initialize output
    torch::Tensor output = torch::zeros_like(input);
    //std::cout << output.sizes() << std::endl;

    const int threadnum_x = min(width,1024);
    std::cout << "threadnum x: " << threadnum_x << std::endl;
    const int threadnum_y = min(height,1024);
    std::cout << "threadnum y: " << threadnum_y << std::endl;
    const dim3 block(num_channel,threadnum_x,threadnum_y);
    std::cout << "block.x: " << block.x << std::endl;
    std::cout << "block.y: " << block.y << std::endl;
    std::cout << "block.z: " << block.z << std::endl;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "nl_relu_gpu", ([&] {
      nl_relu_kernel<scalar_t><<<num_batch, block>>>(
          input.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
          output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>());
    }));

    return {output};
}

// max pooling test kernel
template <typename scalar_t>
__global__ void nl_maxpooling_kernel(
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> input,
    const int pw, const int ph, const int stride_x, const int stride_y,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> output) {
  //batch index
  int n = blockIdx.x;
  //channel index
  int c = threadIdx.x;
  //height index
  int w = threadIdx.y;
  //width index
  int h = threadIdx.z;
  //iteration counter
  int ii, jj;
  if (n < output.size(0) && c < output.size(1) && w < output.size(2) && h < output.size(3)){
    //initialize pooling
    output[n][c][w][h] = input[n][c][w*stride_x][h*stride_y];
    for( ii = w*stride_x; ii < w*stride_x+pw; ii++){
      for( jj = h*stride_y; jj < h*stride_y+ph; jj++){
        if(input[n][c][ii][jj] > output[n][c][w][h]){
          output[n][c][w][h] = input[n][c][ii][jj];
        }
      }
    }
  }
}

// max pooling test kernel wrapper function

std::vector<torch::Tensor> nl_maxpooling_cuda(
  torch::Tensor input,
  torch::Tensor poolsize,
  torch::Tensor stride){
    auto input_size = input.sizes();
    const int num_batch   = input_size[0];
    const int num_channel = input_size[1];
    const int width = input_size[2];
    const int height = input_size[3];
    auto poolsize_a = poolsize.accessor<float,1>();
    auto stride_a = stride.accessor<float,1>();

    std::cout << "Pool window size: (" << poolsize_a[0] << ", " << poolsize_a[1] << ")" << std::endl;
    std::cout << "Stride size: (" << stride_a[0] << ", " << stride_a[1] << ")" << std::endl;

    //std::cout << "(N,C,H,W) = (" << num_batch << ", " << num_channel << ", "
    //<< height << ", " << width << ")" << std::endl;

    //calculate output size
    const int out_h = floor((height-poolsize_a[0])/stride_a[0])+1;
    const int out_w = floor((width-poolsize_a[1])/stride_a[1])+1;
    std::cout << "(out_h,out_w) = (" << out_h << ", " << out_w << ")" << std::endl;
    //ininitalize output
    torch::Tensor output = torch::zeros({num_batch, num_channel, out_w, out_h},
      torch::TensorOptions().device(torch::kCUDA));
    std::cout << output.sizes() << std::endl;

    const int threadnum_x = min(out_w,1024);
    std::cout << "threadnum x: " << threadnum_x << std::endl;
    const int threadnum_y = min(out_h,1024);
    std::cout << "threadnum y: " << threadnum_y << std::endl;
    const dim3 block(num_channel,threadnum_x,threadnum_y);
    std::cout << "block.x: " << block.x << std::endl;
    std::cout << "block.y: " << block.y << std::endl;
    std::cout << "block.z: " << block.z << std::endl;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "nl_maxpool_gpu", ([&] {
      nl_maxpooling_kernel<scalar_t><<<num_batch, block>>>(
          input.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
          poolsize_a[0], poolsize_a[1], stride_a[0], stride_a[1],
          output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>());
    }));

    return {output};
}

// max pooling test kernel
template <typename scalar_t>
__global__ void nl_avgpooling_kernel(
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> input,
    const int pw, const int ph, const int stride_x, const int stride_y,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> output) {
  //batch index
  int n = blockIdx.x;
  //channel index
  int c = threadIdx.x;
  //height index
  int w = threadIdx.y;
  //width index
  int h = threadIdx.z;
  //iteration counter
  int ii, jj;
  if (n < output.size(0) && c < output.size(1) && w < output.size(2) && h < output.size(3)){
    //initialize pooling
    output[n][c][w][h] = 0.0;
    for( ii = w*stride_x; ii < w*stride_x+pw; ii++){
      for( jj = h*stride_y; jj < h*stride_y+ph; jj++){
        output[n][c][w][h] += input[n][c][ii][jj];
      }
    }
    output[n][c][w][h] = output[n][c][w][h]/(pw*ph);
  }
}

// average pooling test kernel wrapper function

std::vector<torch::Tensor> nl_avgpooling_cuda(
  torch::Tensor input,
  torch::Tensor poolsize,
  torch::Tensor stride){
    auto input_size = input.sizes();
    const int num_batch   = input_size[0];
    const int num_channel = input_size[1];
    const int width = input_size[2];
    const int height = input_size[3];
    auto poolsize_a = poolsize.accessor<float,1>();
    auto stride_a = stride.accessor<float,1>();

    std::cout << "Pool window size: (" << poolsize_a[0] << ", " << poolsize_a[1] << ")" << std::endl;
    std::cout << "Stride size: (" << stride_a[0] << ", " << stride_a[1] << ")" << std::endl;

    //std::cout << "(N,C,H,W) = (" << num_batch << ", " << num_channel << ", "
    //<< height << ", " << width << ")" << std::endl;

    //calculate output size
    const int out_h = floor((height-poolsize_a[0])/stride_a[0])+1;
    const int out_w = floor((width-poolsize_a[1])/stride_a[1])+1;
    std::cout << "(out_h,out_w) = (" << out_h << ", " << out_w << ")" << std::endl;
    //ininitalize output
    torch::Tensor output = torch::zeros({num_batch, num_channel, out_w, out_h},
      torch::TensorOptions().device(torch::kCUDA));
    std::cout << output.sizes() << std::endl;

    const int threadnum_x = min(out_w,1024);
    std::cout << "threadnum x: " << threadnum_x << std::endl;
    const int threadnum_y = min(out_h,1024);
    std::cout << "threadnum y: " << threadnum_y << std::endl;
    const dim3 block(num_channel,threadnum_x,threadnum_y);
    std::cout << "block.x: " << block.x << std::endl;
    std::cout << "block.y: " << block.y << std::endl;
    std::cout << "block.z: " << block.z << std::endl;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "nl_maxpool_gpu", ([&] {
      nl_avgpooling_kernel<scalar_t><<<num_batch, block>>>(
          input.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
          poolsize_a[0], poolsize_a[1], stride_a[0], stride_a[1],
          output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>());
    }));

    return {output};
}

// forward kernel function

// forward kernel wrapper function
/*
std::vector<torch::Tensor> lltm_cuda_forward(
  torch::Tensor input,
  torch::Tensor weights,
  torch::Tensor bias,
  torch::Tensor old_h,
  torch::Tensor old_cell){}
*/